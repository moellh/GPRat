#include "hip/hip_runtime.h"
#include "gp_algorithms_gpu.cuh"

#include "cuda_kernels.cuh"
#include "cuda_utils.cuh"
#include "gp_kernels.hpp"
#include "gp_optimizer_gpu.cuh"
#include "target.hpp"
#include "tiled_algorithms_gpu.cuh"
#include <hip/hip_runtime.h>
#include <hpx/algorithm.hpp>
#include <hpx/async_cuda/cuda_exception.hpp>

#if GPRAT_CHOLESKY_STEPS || GPRAT_ASSEMBLY_ONLY
    #include <apex_api.hpp>
#endif

using hpx::experimental::for_loop;

namespace gpu
{

// Kernel function to compute covariance
__global__ void
gen_tile_covariance_kernel(double *d_tile,
                           const double *d_input,
                           const std::size_t n_tile_size,
                           const std::size_t n_regressors,
                           const std::size_t tile_row,
                           const std::size_t tile_column,
                           const gpxpy_hyper::SEKParams sek_params)
{
    // Compute the global indices of the thread
    unsigned int i = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < n_tile_size && j < n_tile_size)
    {
        std::size_t i_global = n_tile_size * tile_row + i;
        std::size_t j_global = n_tile_size * tile_column + j;

        double distance = 0.0;
        for (std::size_t k = 0; k < n_regressors; ++k)
        {
            int offset = -n_regressors + 1 + k;
            int i_local = i_global + offset;
            int j_local = j_global + offset;

            double z_ik = (i_local >= 0) ? d_input[i_local] : 0.0;
            double z_jk = (j_local >= 0) ? d_input[j_local] : 0.0;
            distance += (z_ik - z_jk) * (z_ik - z_jk);
        }

        // Compute the covariance value
        double covariance = sek_params.vertical_lengthscale * exp(-0.5 * distance / (sek_params.lengthscale * sek_params.lengthscale));

        // Add noise variance if diagonal
        if (i_global == j_global)
        {
            covariance += sek_params.noise_variance;
        }

        d_tile[i * n_tile_size + j] = covariance;
    }
}

double *
gen_tile_covariance(const double *d_input,
                    const std::size_t tile_row,
                    const std::size_t tile_column,
                    const std::size_t n_tile_size,
                    const std::size_t n_regressors,
                    const gpxpy_hyper::SEKParams sek_params,
                    gpxpy::CUDA_GPU &gpu)
{
    double *d_tile;

    dim3 threads_per_block(16, 16);
    dim3 n_blocks((n_tile_size + 15) / 16, (n_tile_size + 15) / 16);

    hipStream_t stream = gpu.next_stream();

    check_cuda_error(hipMalloc(&d_tile, n_tile_size * n_tile_size * sizeof(double)));
    gen_tile_covariance_kernel<<<n_blocks, threads_per_block, gpu.shared_memory_size, stream>>>(d_tile, d_input, n_tile_size, n_regressors, tile_row, tile_column, sek_params);

    check_cuda_error(hipStreamSynchronize(stream));

    return d_tile;
}

__global__ void
gen_tile_full_prior_covariance_kernel(double *d_tile,
                                      const double *d_input,
                                      const std::size_t n_tile_size,
                                      const std::size_t n_regressors,
                                      const std::size_t tile_row,
                                      const std::size_t tile_column,
                                      const gpxpy_hyper::SEKParams sek_params)
{
    unsigned int i = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < n_tile_size && j < n_tile_size)
    {
        std::size_t i_global = n_tile_size * tile_row + i;
        std::size_t j_global = n_tile_size * tile_column + j;

        double distance = 0.0;
        for (std::size_t k = 0; k < n_regressors; ++k)
        {
            int offset = -n_regressors + 1 + k;
            int i_local = i_global + offset;
            int j_local = j_global + offset;

            double z_ik = (i_local >= 0) ? d_input[i_local] : 0.0;
            double z_jk = (j_local >= 0) ? d_input[j_local] : 0.0;
            distance += (z_ik - z_jk) * (z_ik - z_jk);
        }

        double covariance = sek_params.vertical_lengthscale * exp(-0.5 * distance / (sek_params.lengthscale * sek_params.lengthscale));

        d_tile[i * n_tile_size + j] = covariance;
    }
}

double *
gen_tile_full_prior_covariance(const double *d_input,
                               const std::size_t tile_row,
                               const std::size_t tile_colums,
                               const std::size_t n_tile_size,
                               const std::size_t n_regressors,
                               const gpxpy_hyper::SEKParams sek_params,
                               gpxpy::CUDA_GPU &gpu)
{
    double *d_tile;

    dim3 threads_per_block(16, 16);
    dim3 n_blocks((n_tile_size + 15) / 16, (n_tile_size + 15) / 16);

    hipStream_t stream = gpu.next_stream();

    check_cuda_error(hipMalloc(&d_tile, n_tile_size * n_tile_size * sizeof(double)));
    gen_tile_full_prior_covariance_kernel<<<n_blocks, threads_per_block, gpu.shared_memory_size, stream>>>(d_tile, d_input, n_tile_size, n_regressors, tile_row, tile_colums, sek_params);

    check_cuda_error(hipStreamSynchronize(stream));

    return d_tile;
}

__global__ void
gen_tile_prior_covariance_kernel(double *d_tile,
                                 const double *d_input,
                                 const std::size_t n_tile_size,
                                 const std::size_t n_regressors,
                                 const std::size_t tile_row,
                                 const std::size_t tile_column,
                                 const gpxpy_hyper::SEKParams sek_params)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < n_tile_size)
    {
        std::size_t i_global = n_tile_size * tile_row + i;
        std::size_t j_global = n_tile_size * tile_column + i;

        double distance = 0.0;
        for (std::size_t k = 0; k < n_regressors; ++k)
        {
            int offset = -n_regressors + 1 + k;
            int i_local = i_global + offset;
            int j_local = j_global + offset;

            double z_ik = (i_local >= 0) ? d_input[i_local] : 0.0;
            double z_jk = (j_local >= 0) ? d_input[j_local] : 0.0;
            distance += (z_ik - z_jk) * (z_ik - z_jk);
        }

        double covariance = sek_params.vertical_lengthscale * exp(-0.5 * distance / (sek_params.lengthscale * sek_params.lengthscale));

        d_tile[i] = covariance;
    }
}

double *
gen_tile_prior_covariance(const double *d_input,
                          const std::size_t tile_row,
                          const std::size_t tile_column,
                          const std::size_t n_tile_size,
                          const std::size_t n_regressors,
                          const gpxpy_hyper::SEKParams sek_params,
                          gpxpy::CUDA_GPU &gpu)
{
    double *d_tile;

    dim3 threads_per_block(256);
    dim3 n_blocks((n_tile_size + 255) / 256);

    hipStream_t stream = gpu.next_stream();

    check_cuda_error(hipMalloc(&d_tile, n_tile_size * sizeof(double)));
    gen_tile_prior_covariance_kernel<<<n_blocks, threads_per_block, 0, stream>>>(d_tile, d_input, n_tile_size, n_regressors, tile_row, tile_column, sek_params);

    check_cuda_error(hipStreamSynchronize(stream));

    return d_tile;
}

__global__ void
gen_tile_cross_covariance_kernel(double *d_tile,
                                 const double *d_row_input,
                                 const double *d_col_input,
                                 const std::size_t n_row_tile_size,
                                 const std::size_t n_column_tile_size,
                                 const std::size_t tile_row,
                                 const std::size_t tile_column,
                                 const std::size_t n_regressors,
                                 const gpxpy_hyper::SEKParams sek_params)
{
    unsigned int i = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < n_row_tile_size && j < n_column_tile_size)
    {
        std::size_t i_global = n_row_tile_size * tile_row + i;
        std::size_t j_global = n_column_tile_size * tile_column + j;

        double z_ik = 0.0;
        double z_jk = 0.0;
        double distance = 0.0;

        for (std::size_t k = 0; k < n_regressors; ++k)
        {
            int offset = -n_regressors + 1 + k;
            int i_local = i_global + offset;
            int j_local = j_global + offset;

            if (i_local >= 0)
            {
                z_ik = d_row_input[i_local];
            }
            if (j_local >= 0)
            {
                z_jk = d_col_input[j_local];
            }
            distance += (z_ik - z_jk) * (z_ik - z_jk);
        }

        double covariance = sek_params.vertical_lengthscale * exp(-0.5 * distance / (sek_params.lengthscale * sek_params.lengthscale));
        d_tile[i * n_column_tile_size + j] = covariance;
    }
}

double *
gen_tile_cross_covariance(const double *d_row_input,
                          const double *d_col_input,
                          const std::size_t tile_row,
                          const std::size_t tile_column,
                          const std::size_t n_row_tile_size,
                          const std::size_t n_column_tile_size,
                          const std::size_t n_regressors,
                          const gpxpy_hyper::SEKParams sek_params,
                          gpxpy::CUDA_GPU &gpu)
{
    double *d_tile;

    dim3 threads_per_block(16, 16);
    dim3 n_blocks((n_column_tile_size + 15) / 16, (n_row_tile_size + 15) / 16);

    hipStream_t stream = gpu.next_stream();

    check_cuda_error(hipMalloc(&d_tile, n_row_tile_size * n_column_tile_size * sizeof(double)));
    gen_tile_cross_covariance_kernel<<<n_blocks, threads_per_block, 0, stream>>>(d_tile, d_row_input, d_col_input, n_row_tile_size, n_column_tile_size, tile_row, tile_column, n_regressors, sek_params);

    check_cuda_error(hipStreamSynchronize(stream));

    return d_tile;
}

hpx::shared_future<double *>
gen_tile_cross_cov_T(
    std::size_t n_row_tile_size,
    std::size_t n_column_tile_size,
    const hpx::shared_future<double *> f_cross_covariance_tile,
    gpxpy::CUDA_GPU &gpu)
{
    double *transposed;
    check_cuda_error(hipMalloc(&transposed, n_row_tile_size * n_column_tile_size * sizeof(double)));
    double *d_cross_covariance_tile = f_cross_covariance_tile.get();

    hipStream_t stream = gpu.next_stream();
    dim3 threads_per_block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 n_blocks((n_column_tile_size + BLOCK_SIZE - 1) / BLOCK_SIZE, (n_row_tile_size + BLOCK_SIZE - 1) / BLOCK_SIZE);

    transpose<<<n_blocks, threads_per_block, 0, stream>>>(transposed, d_cross_covariance_tile, n_row_tile_size, n_column_tile_size);

    check_cuda_error(hipStreamSynchronize(stream));

    return hpx::make_ready_future(transposed);
}

__global__ void
gen_tile_output_kernel(double *tile,
                       const double *output,
                       std::size_t row,
                       std::size_t n_tile_size)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n_tile_size)
    {
        std::size_t i_global = n_tile_size * row + i;
        tile[i] = output[i_global];
    }
}

double *
gen_tile_output(const std::size_t row,
                const std::size_t n_tile_size,
                const double *d_output,
                gpxpy::CUDA_GPU &gpu)
{
    dim3 threads_per_block(256);
    dim3 n_blocks((n_tile_size + 255) / 256);

    hipStream_t stream = gpu.next_stream();

    double *d_tile;
    check_cuda_error(hipMalloc(&d_tile, n_tile_size * sizeof(double)));

    gen_tile_output_kernel<<<n_blocks, threads_per_block, 0, stream>>>(d_tile, d_output, row, n_tile_size);

    check_cuda_error(hipStreamSynchronize(stream));

    return d_tile;
}

double *gen_tile_zeros(std::size_t n_tile_size, gpxpy::CUDA_GPU &gpu)
{
    double *d_tile;
    hipStream_t stream = gpu.next_stream();
    check_cuda_error(hipMalloc(&d_tile, n_tile_size * sizeof(double)));
    check_cuda_error(hipMemsetAsync(d_tile, 0, n_tile_size * sizeof(double), stream));
    check_cuda_error(hipStreamSynchronize(stream));
    return d_tile;
}

double compute_error_norm(std::size_t n_tiles,
                          std::size_t n_tile_size,
                          const std::vector<double> &b,
                          const std::vector<std::vector<double>> &tiles)
{
    double error = 0.0;
    for (std::size_t k = 0; k < n_tiles; k++)
    {
        auto a = tiles[k];
        for (std::size_t i = 0; i < n_tile_size; i++)
        {
            std::size_t i_global = n_tile_size * k + i;
            // ||a - b||_2
            error += (b[i_global] - a[i]) * (b[i_global] - a[i]);
        }
    }
    return sqrt(error);
}

std::vector<hpx::shared_future<double *>>
assemble_tiled_covariance_matrix(const double *d_training_input,
                                 const std::size_t n_tiles,
                                 const std::size_t n_tile_size,
                                 const std::size_t n_regressors,
                                 const gpxpy_hyper::SEKParams sek_params,
                                 gpxpy::CUDA_GPU &gpu)
{
    std::vector<hpx::shared_future<double *>> d_tiles(n_tiles * n_tiles);

    for (std::size_t tile_row = 0; tile_row < n_tiles; ++tile_row)
    {
        for (std::size_t tile_column = 0; tile_column <= tile_row; ++tile_column)
        {
            d_tiles[tile_row * n_tiles + tile_column] = hpx::async(
                hpx::annotated_function(&gen_tile_covariance, "assemble K_tiles"),
                d_training_input,
                tile_row,
                tile_column,
                n_tile_size,
                n_regressors,
                sek_params,
                std::ref(gpu));
        }
    }

    return d_tiles;
}

std::vector<hpx::shared_future<double *>>
assemble_alpha_tiles(const double *d_output,
                     const std::size_t n_tiles,
                     const std::size_t n_tile_size,
                     gpxpy::CUDA_GPU &gpu)
{
    std::vector<hpx::shared_future<double *>> alpha_tiles(n_tiles);
    for (std::size_t i = 0; i < n_tiles; i++)
    {
        alpha_tiles[i] = hpx::async(hpx::annotated_function(&gen_tile_output, "assemble alpha_tiles"), i, n_tile_size, d_output, std::ref(gpu));
    }

    return alpha_tiles;
}

std::vector<hpx::shared_future<double *>>
assemble_cross_covariance_tiles(const double *d_test_input,
                                const double *d_training_input,
                                const std::size_t m_tiles,
                                const std::size_t n_tiles,
                                const std::size_t m_tile_size,
                                const std::size_t n_tile_size,
                                const std::size_t n_regressors,
                                const gpxpy_hyper::SEKParams sek_params,
                                gpxpy::CUDA_GPU &gpu)
{
    std::vector<hpx::shared_future<double *>> cross_covariance_tiles;
    cross_covariance_tiles.resize(m_tiles * n_tiles);
    for (std::size_t i = 0; i < m_tiles; i++)
    {
        for (std::size_t j = 0; j < n_tiles; j++)
        {
            cross_covariance_tiles[i * n_tiles + j] = hpx::async(hpx::annotated_function(&gen_tile_cross_covariance, "assemble cross_covariance_tiles"), d_test_input, d_training_input, i, j, m_tile_size, n_tile_size, n_regressors, sek_params, std::ref(gpu));
        }
    }
    return cross_covariance_tiles;
}

std::vector<hpx::shared_future<double *>>
assemble_tiles_with_zeros(std::size_t n_tile_size, std::size_t n_tiles, gpxpy::CUDA_GPU &gpu)
{
    std::vector<hpx::shared_future<double *>> tiles(n_tiles);
    for (std::size_t i = 0; i < n_tiles; i++)
    {
        tiles[i] = hpx::async(
            hpx::annotated_function(&gen_tile_zeros, "assemble prediction_tiles"),
            n_tile_size,
            std::ref(gpu));
    }
    return tiles;
}

std::vector<hpx::shared_future<double *>>
assemble_prior_K_tiles(const double *d_test_input,
                       const std::size_t m_tiles,
                       const std::size_t m_tile_size,
                       const std::size_t n_regressors,
                       const gpxpy_hyper::SEKParams sek_params,
                       gpxpy::CUDA_GPU &gpu)
{
    std::vector<hpx::shared_future<double *>> d_prior_K_tiles;
    d_prior_K_tiles.resize(m_tiles);
    for (std::size_t i = 0; i < m_tiles; i++)
    {
        d_prior_K_tiles[i] = hpx::async(
            &gen_tile_prior_covariance,
            d_test_input,
            i,
            i,
            m_tile_size,
            n_regressors,
            sek_params,
            std::ref(gpu));
    }
    return d_prior_K_tiles;
}

std::vector<hpx::shared_future<double *>>
assemble_prior_K_tiles_full(const double *d_test_input,
                            const std::size_t m_tiles,
                            const std::size_t m_tile_size,
                            const std::size_t n_regressors,
                            const gpxpy_hyper::SEKParams sek_params,
                            gpxpy::CUDA_GPU &gpu)
{
    std::vector<hpx::shared_future<double *>> d_prior_K_tiles(m_tiles * m_tiles);
    for (std::size_t i = 0; i < m_tiles; i++)
    {
        for (std::size_t j = 0; j <= i; j++)
        {
            d_prior_K_tiles[i * m_tiles + j] = hpx::async(
                &gen_tile_full_prior_covariance,
                d_test_input,
                i,
                j,
                m_tile_size,
                n_regressors,
                sek_params,
                std::ref(gpu));

            if (i != j)
            {
                d_prior_K_tiles[j * m_tiles + i] = hpx::dataflow(
                    &gen_tile_grad_l_trans,
                    m_tile_size,
                    d_prior_K_tiles[i * m_tiles + j],
                    std::ref(gpu));
            }
        }
    }
    return d_prior_K_tiles;
}

std::vector<hpx::shared_future<double *>>
assemble_t_cross_covariance_tiles(
    const std::vector<hpx::shared_future<double *>> &d_cross_covariance_tiles,
    const std::size_t n_tiles,
    const std::size_t m_tiles,
    const std::size_t n_tile_size,
    const std::size_t m_tile_size,
    gpxpy::CUDA_GPU &gpu)
{
    std::vector<hpx::shared_future<double *>> d_t_cross_covariance_tiles(m_tiles * n_tiles);
    for (std::size_t i = 0; i < m_tiles; i++)
    {
        for (std::size_t j = 0; j < n_tiles; j++)
        {
            d_t_cross_covariance_tiles[j * m_tiles + i] = hpx::dataflow(
                &gen_tile_cross_cov_T,
                m_tile_size,
                n_tile_size,
                d_cross_covariance_tiles[i * n_tiles + j],
                std::ref(gpu));
        }
    }
    return d_t_cross_covariance_tiles;
}

std::vector<hpx::shared_future<double *>>
assemble_y_tiles(
    const double *d_training_output,
    const std::size_t n_tiles,
    const std::size_t n_tile_size,
    gpxpy::CUDA_GPU &gpu)
{
    std::vector<hpx::shared_future<double *>> d_y_tiles(n_tiles);
    for (std::size_t i = 0; i < n_tiles; i++)
    {
        d_y_tiles[i] = hpx::async(&gen_tile_output, i, n_tile_size, d_training_output, std::ref(gpu));
    }
    return d_y_tiles;
}

std::vector<double>
copy_tiled_vector_to_host_vector(
    std::vector<hpx::shared_future<double *>> &d_tiles,
    std::size_t n_tile_size,
    std::size_t n_tiles,
    gpxpy::CUDA_GPU &gpu)
{
    std::vector<double> h_vector(n_tiles * n_tile_size);
    std::vector<hipStream_t> streams(n_tiles);
    for (std::size_t i = 0; i < n_tiles; i++)
    {
        streams[i] = gpu.next_stream();
        check_cuda_error(hipMemcpyAsync(h_vector.data() + i * n_tile_size, d_tiles[i].get(), n_tile_size * sizeof(double), hipMemcpyDeviceToHost, streams[i]));
    }
    gpu.sync_streams(streams);
    return h_vector;
}

std::vector<std::vector<double>>
move_lower_tiled_matrix_to_host(
    const std::vector<hpx::shared_future<double *>> &d_tiles,
    const std::size_t n_tile_size,
    const std::size_t n_tiles,
    gpxpy::CUDA_GPU &gpu)
{
    std::vector<std::vector<double>> h_tiles(n_tiles * n_tiles);

    std::vector<hipStream_t> streams(n_tiles * (n_tiles + 1) / 2);
    for (std::size_t i = 0; i < n_tiles; ++i)
    {
        for (std::size_t j = 0; j <= i; ++j)
        {
            streams[i] = gpu.next_stream();
            h_tiles[i * n_tiles + j].resize(n_tile_size * n_tile_size);
            check_cuda_error(hipMemcpyAsync(h_tiles[i * n_tiles + j].data(), d_tiles[i * n_tiles + j].get(), n_tile_size * n_tile_size * sizeof(double), hipMemcpyDeviceToHost, streams[i]));
            check_cuda_error(hipFree(d_tiles[i * n_tiles + j].get()));
        }
    }
    gpu.sync_streams(streams);

    return h_tiles;
}

void free_lower_tiled_matrix(
    const std::vector<hpx::shared_future<double *>> &d_tiles,
    const std::size_t n_tiles)
{
    for (std::size_t i = 0; i < n_tiles; ++i)
    {
        for (std::size_t j = 0; j <= i; ++j)
        {
            check_cuda_error(hipFree(d_tiles[i * n_tiles + j].get()));
        }
    }
}

hpx::shared_future<std::vector<double>>
predict(const std::vector<double> &h_training_input,
        const std::vector<double> &h_training_output,
        const std::vector<double> &h_test_input,
        const std::size_t n_tiles,
        const std::size_t n_tile_size,
        const std::size_t m_tiles,
        const std::size_t m_tile_size,
        const std::size_t n_regressors,
        const gpxpy_hyper::SEKParams sek_params,
        gpxpy::CUDA_GPU &gpu)
{
#if GPRAT_PREDICT_STEPS
    auto predict_step_ra_timer = apex::start("predict_step ressource allocation");
#endif
    gpu.create();
    hipsolverHandle_t cusolver = create_cusolver_handle();
#if GPRAT_PREDICT_STEPS
    apex::stop(predict_step_ra_timer);
    auto predict_step_assembly_timer = apex::start("predict_step assembly");
#endif
    double *d_training_input = copy_to_device(h_training_input, gpu);
    double *d_training_output = copy_to_device(h_training_output, gpu);
    double *d_test_input = copy_to_device(h_test_input, gpu);

    auto d_tiles = assemble_tiled_covariance_matrix(d_training_input, n_tiles, n_tile_size, n_regressors, sek_params, gpu);
    auto alpha_tiles = assemble_alpha_tiles(d_training_output, n_tiles, n_tile_size, gpu);
    auto cross_covariance_tiles = assemble_cross_covariance_tiles(d_test_input, d_training_input, m_tiles, n_tiles, m_tile_size, n_tile_size, n_regressors, sek_params, gpu);
    auto prediction_tiles = assemble_tiles_with_zeros(m_tile_size, m_tiles, gpu);

#if GPRAT_PREDICT_STEPS
    hpx::wait_all(d_tiles, alpha_tiles, cross_covariance_tiles, prediction_tiles);
    apex::stop(predict_step_assembly_timer);
    auto predict_step_cholesky_timer = apex::start("predict_step cholesky");
#endif

    right_looking_cholesky_tiled(d_tiles, n_tile_size, n_tiles, gpu, cusolver);

#if GPRAT_PREDICT_STEPS
    hpx::wait_all(d_tiles);
    apex::stop(predict_step_cholesky_timer);
    auto predict_step_forward_timer = apex::start("predict_step forward");
#endif

    // Triangular solve K_NxN * alpha = y
    forward_solve_tiled(d_tiles, alpha_tiles, n_tile_size, n_tiles, gpu);

#if GPRAT_PREDICT_STEPS
    hpx::wait_all(alpha_tiles);
    apex::stop(predict_step_forward_timer);
    auto predict_step_backward_timer = apex::start("predict_step backward");
#endif

    backward_solve_tiled(d_tiles, alpha_tiles, n_tile_size, n_tiles, gpu);

#if GPRAT_PREDICT_STEPS
    hpx::wait_all(alpha_tiles);
    apex::stop(predict_step_backward_timer);
    auto predict_step_prediction_timer = apex::start("predict_step prediction");
#endif

    prediction_tiled(cross_covariance_tiles, alpha_tiles, prediction_tiles, m_tile_size, n_tile_size, n_tiles, m_tiles, gpu);

#if GPRAT_PREDICT_STEPS
    hpx::wait_all(prediction_tiles);
    apex::stop(predict_step_prediction_timer);
    auto predict_step_copyback_timer = apex::start("predict_step copyback");
#endif
    std::vector<double> prediction = copy_tiled_vector_to_host_vector(prediction_tiles, m_tile_size, m_tiles, gpu);

#if GPRAT_PREDICT_STEPS
    apex::stop(predict_step_copyback_timer);
    auto predict_step_rd_timer = apex::start("cholesky_step ressource destroy");
#endif
    free_lower_tiled_matrix(d_tiles, n_tiles);
    free(alpha_tiles);
    free(cross_covariance_tiles);
    free(prediction_tiles);
    destroy(cusolver);

    gpu.destroy();

#if GPRAT_PREDICT_STEPS
    apex::stop(predict_step_rd_timer);
#endif

    return hpx::make_ready_future(prediction);
}

hpx::shared_future<std::vector<std::vector<double>>>
predict_with_uncertainty(const std::vector<double> &h_training_input,
                         const std::vector<double> &h_training_output,
                         const std::vector<double> &h_test_input,
                         const std::size_t n_tiles,
                         const std::size_t n_tile_size,
                         const std::size_t m_tiles,
                         const std::size_t m_tile_size,
                         const std::size_t n_regressors,
                         const gpxpy_hyper::SEKParams sek_params,
                         gpxpy::CUDA_GPU &gpu)
{
    gpu.create();

    double *d_training_input = copy_to_device(h_training_input, gpu);
    double *d_training_output = copy_to_device(h_training_output, gpu);
    double *d_test_input = copy_to_device(h_test_input, gpu);

    // Assemble tiled covariance matrix on GPU.
    auto d_K_tiles = assemble_tiled_covariance_matrix(d_training_input, n_tiles, n_tile_size, n_regressors, sek_params, gpu);

    auto d_alpha_tiles = assemble_alpha_tiles(d_training_output, n_tiles, n_tile_size, gpu);

    auto d_prior_K_tiles = assemble_prior_K_tiles(d_test_input, m_tiles, m_tile_size, n_regressors, sek_params, gpu);

    auto d_cross_covariance_tiles = assemble_cross_covariance_tiles(d_test_input, d_training_input, m_tiles, n_tiles, m_tile_size, n_tile_size, n_regressors, sek_params, gpu);

    auto d_t_cross_covariance_tiles = assemble_t_cross_covariance_tiles(d_cross_covariance_tiles, n_tiles, m_tiles, n_tile_size, m_tile_size, gpu);

    // Assemble placeholder matrix for diag(K_MxN * (K^-1_NxN * K_NxM))
    auto d_prior_inter_tiles = assemble_tiles_with_zeros(m_tile_size, m_tiles, gpu);

    auto d_prediction_tiles = assemble_tiles_with_zeros(m_tile_size, m_tiles, gpu);

    // Assemble placeholder for uncertainty
    auto d_prediction_uncertainty_tiles = assemble_tiles_with_zeros(m_tile_size, m_tiles, gpu);

    hipsolverHandle_t cusolver = create_cusolver_handle();
    right_looking_cholesky_tiled(d_K_tiles, n_tile_size, n_tiles, gpu, cusolver);

    // Triangular solve K_NxN * alpha = y
    forward_solve_tiled(d_K_tiles, d_alpha_tiles, n_tile_size, n_tiles, gpu);
    backward_solve_tiled(d_K_tiles, d_alpha_tiles, n_tile_size, n_tiles, gpu);

    // Triangular solve A_M,N * K_NxN = K_MxN -> A_MxN = K_MxN * K^-1_NxN
    forward_solve_KcK_tiled(d_K_tiles, d_t_cross_covariance_tiles, n_tile_size, m_tile_size, n_tiles, m_tiles, gpu);

    // Compute predictions
    prediction_tiled(d_cross_covariance_tiles, d_alpha_tiles, d_prediction_tiles, m_tile_size, n_tile_size, n_tiles, m_tiles, gpu);

    // posterior covariance matrix - (K_MxN * K^-1_NxN) * K_NxM
    posterior_covariance_tiled(d_t_cross_covariance_tiles, d_prior_inter_tiles, n_tile_size, m_tile_size, n_tiles, m_tiles, gpu);

    // Compute predicition uncertainty
    prediction_uncertainty_tiled(d_prior_K_tiles, d_prior_inter_tiles, d_prediction_uncertainty_tiles, m_tile_size, m_tiles, gpu);

    // Get predictions and uncertainty to return them
    std::vector<double> prediction = copy_tiled_vector_to_host_vector(d_prediction_tiles, m_tile_size, m_tiles, gpu);
    std::vector<double> pred_var_full = copy_tiled_vector_to_host_vector(d_prediction_uncertainty_tiles, m_tile_size, m_tiles, gpu);

    check_cuda_error(hipFree(d_training_input));
    check_cuda_error(hipFree(d_training_output));
    check_cuda_error(hipFree(d_test_input));
    free_lower_tiled_matrix(d_K_tiles, n_tiles);
    free(d_alpha_tiles);
    free(d_prior_K_tiles);
    free(d_cross_covariance_tiles);
    free(d_t_cross_covariance_tiles);
    free(d_prior_inter_tiles);
    free(d_prediction_tiles);
    free(d_prediction_uncertainty_tiles);
    destroy(cusolver);

    gpu.destroy();

    return hpx::make_ready_future(std::vector<std::vector<double>>{ prediction, pred_var_full });
}

hpx::shared_future<std::vector<std::vector<double>>>
predict_with_full_cov(const std::vector<double> &h_training_input,
                      const std::vector<double> &h_training_output,
                      const std::vector<double> &h_test_input,
                      int n_tiles,
                      int n_tile_size,
                      int m_tiles,
                      int m_tile_size,
                      int n_regressors,
                      gpxpy_hyper::SEKParams sek_params,
                      gpxpy::CUDA_GPU &gpu)
{
    gpu.create();

    double *d_training_input = copy_to_device(h_training_input, gpu);
    double *d_training_output = copy_to_device(h_training_output, gpu);
    double *d_test_input = copy_to_device(h_test_input, gpu);

    // Assemble tiled covariance matrix on GPU.
    auto d_K_tiles = assemble_tiled_covariance_matrix(d_training_input, n_tiles, n_tile_size, n_regressors, sek_params, gpu);

    auto d_alpha_tiles = assemble_alpha_tiles(d_training_output, n_tiles, n_tile_size, gpu);

    auto d_prior_K_tiles = assemble_prior_K_tiles_full(d_test_input, m_tiles, m_tile_size, n_regressors, sek_params, gpu);

    auto d_cross_covariance_tiles = assemble_cross_covariance_tiles(d_test_input, d_training_input, m_tiles, n_tiles, m_tile_size, n_tile_size, n_regressors, sek_params, gpu);

    auto d_t_cross_covariance_tiles = assemble_t_cross_covariance_tiles(d_cross_covariance_tiles, n_tiles, m_tiles, n_tile_size, m_tile_size, gpu);

    // Assemble placeholder matrix for diag(K_MxN * (K^-1_NxN * K_NxM))
    auto d_prior_inter_tiles = assemble_tiles_with_zeros(m_tile_size, m_tiles, gpu);

    auto d_prediction_tiles = assemble_tiles_with_zeros(m_tile_size, m_tiles, gpu);

    // Assemble placeholder for uncertainty
    auto d_prediction_uncertainty_tiles = assemble_tiles_with_zeros(m_tile_size, m_tiles, gpu);

    hipsolverHandle_t cusolver = create_cusolver_handle();
    right_looking_cholesky_tiled(d_K_tiles, n_tile_size, n_tiles, gpu, cusolver);

    // Triangular solve K_NxN * alpha = y
    forward_solve_tiled(d_K_tiles, d_alpha_tiles, n_tile_size, n_tiles, gpu);
    backward_solve_tiled(d_K_tiles, d_alpha_tiles, n_tile_size, n_tiles, gpu);

    // Triangular solve A_M,N * K_NxN = K_MxN -> A_MxN = K_MxN * K^-1_NxN
    forward_solve_KcK_tiled(d_K_tiles, d_t_cross_covariance_tiles, n_tile_size, m_tile_size, n_tiles, m_tiles, gpu);

    // Compute predictions
    prediction_tiled(d_cross_covariance_tiles, d_alpha_tiles, d_prediction_tiles, m_tile_size, n_tile_size, n_tiles, m_tiles, gpu);

    // posterior covariance matrix - (K_MxN * K^-1_NxN) * K_NxM
    full_cov_tiled(d_t_cross_covariance_tiles, d_prior_K_tiles, n_tile_size, m_tile_size, n_tiles, m_tiles, gpu);

    // Compute predicition uncertainty
    pred_uncer_tiled(d_prior_K_tiles, d_prediction_uncertainty_tiles, m_tile_size, m_tiles, gpu);

    // Get predictions and uncertainty to return them
    std::vector<double> prediction = copy_tiled_vector_to_host_vector(d_prediction_tiles, m_tile_size, m_tiles, gpu);
    std::vector<double> pred_var_full = copy_tiled_vector_to_host_vector(d_prediction_uncertainty_tiles, m_tile_size, m_tiles, gpu);

    check_cuda_error(hipFree(d_training_input));
    check_cuda_error(hipFree(d_training_output));
    check_cuda_error(hipFree(d_test_input));
    free_lower_tiled_matrix(d_K_tiles, n_tiles);
    free(d_alpha_tiles);
    free_lower_tiled_matrix(d_prior_K_tiles, m_tiles);
    free(d_cross_covariance_tiles);
    free(d_t_cross_covariance_tiles);
    free(d_prior_inter_tiles);
    free(d_prediction_tiles);
    free(d_prediction_uncertainty_tiles);
    destroy(cusolver);

    gpu.destroy();

    return hpx::make_ready_future(std::vector<std::vector<double>>{ prediction, pred_var_full });
}

hpx::shared_future<double>
compute_loss(const std::vector<double> &h_training_input,
             const std::vector<double> &h_training_output,
             const std::size_t n_tiles,
             const std::size_t n_tile_size,
             const std::size_t n_regressors,
             const gpxpy_hyper::SEKParams sek_params,
             gpxpy::CUDA_GPU &gpu)
{
    gpu.create();

    double *d_training_input = copy_to_device(h_training_input, gpu);
    double *d_training_output = copy_to_device(h_training_output, gpu);

    // Assemble tiled covariance matrix on GPU.
    auto d_K_tiles = assemble_tiled_covariance_matrix(d_training_input, n_tiles, n_tile_size, n_regressors, sek_params, gpu);

    auto d_alpha_tiles = assemble_alpha_tiles(d_training_output, n_tiles, n_tile_size, gpu);

    auto d_y_tiles = assemble_y_tiles(d_training_output, n_tiles, n_tile_size, gpu);

    hipsolverHandle_t cusolver = create_cusolver_handle();
    right_looking_cholesky_tiled(d_K_tiles, n_tile_size, n_tiles, gpu, cusolver);

    // Triangular solve K_NxN * alpha = y
    forward_solve_tiled(d_K_tiles, d_alpha_tiles, n_tile_size, n_tiles, gpu);
    backward_solve_tiled(d_K_tiles, d_alpha_tiles, n_tile_size, n_tiles, gpu);

    // Compute loss
    hpx::shared_future<double> loss_value = compute_loss_tiled(d_K_tiles, d_alpha_tiles, d_y_tiles, n_tile_size, n_tiles, gpu);

    check_cuda_error(hipFree(d_training_input));
    check_cuda_error(hipFree(d_training_output));

    loss_value.get();
    free_lower_tiled_matrix(d_K_tiles, n_tiles);
    free(d_alpha_tiles);
    free(d_y_tiles);
    destroy(cusolver);

    gpu.destroy();

    return loss_value;
}

hpx::shared_future<std::vector<double>>
optimize(const std::vector<double> &training_input,
         const std::vector<double> &training_output,
         const std::size_t n_tiles,
         const std::size_t n_tile_size,
         const std::size_t n_regressors,
         const gpxpy_hyper::SEKParams &sek_params,
         const std::vector<bool> trainable_params,
         const gpxpy_hyper::AdamParams &adam_params,
         gpxpy::CUDA_GPU &gpu)
{
    /* // declaretiled future data structures
    std::vector<hpx::shared_future<std::vector<double>>> K_tiles;
    std::vector<hpx::shared_future<std::vector<double>>> grad_v_tiles;
    std::vector<hpx::shared_future<std::vector<double>>> grad_l_tiles;
    std::vector<hpx::shared_future<std::vector<double>>> grad_K_tiles;
    std::vector<hpx::shared_future<std::vector<double>>> grad_I_tiles;
    std::vector<hpx::shared_future<std::vector<double>>> alpha_tiles;
    std::vector<hpx::shared_future<std::vector<double>>> y_tiles;
    // data holders for Adam
    std::vector<hpx::shared_future<double>> m_T;
    std::vector<hpx::shared_future<double>> v_T;
    std::vector<hpx::shared_future<double>> beta1_T;
    std::vector<hpx::shared_future<double>> beta2_T;
    // data holder for loss
    hpx::shared_future<double> loss_value;
    // data holder for computed loss values
    std::vector<double> losses;
    losses.resize(adam_params.opt_iter);
    //////////////////////////////////////////////////////////////////////////////
    // Assemble beta1_t and beta2_t
    beta1_T.resize(adam_params.opt_iter);
    for (int i = 0; i < adam_params.opt_iter; i++)
    {
        beta1_T[i] = hpx::async(
            hpx::annotated_function(&gen_beta_T, "assemble_tiled"), i + 1, adam_params.beta1);
    }
    beta2_T.resize(adam_params.opt_iter);
    for (int i = 0; i < adam_params.opt_iter; i++)
    {
        beta2_T[i] = hpx::async(
            hpx::annotated_function(&gen_beta_T, "assemble_tiled"), i + 1, adam_params.beta2);
    }
    // Assemble first and second momemnt vectors: m_T and v_T
    m_T.resize(3);
    v_T.resize(3);
    for (int i = 0; i < 3; i++)
    {
        m_T[i] = hpx::async(
            hpx::annotated_function(&gen_moment, "assemble_tiled"));
        v_T[i] = hpx::async(
            hpx::annotated_function(&gen_moment, "assemble_tiled"));
    }

    // Assemble y
    y_tiles.resize(n_tiles);
    for (std::size_t i = 0; i < n_tiles; i++)
    {
        y_tiles[i] = hpx::async(
            hpx::annotated_function(&gen_tile_output, "assemble_y"), i, n_tile_size, training_output);
    }

    // Perform optimization
    for (int iter = 0; iter < adam_params.opt_iter; iter++)
    {
        // Assemble covariance matrix vector, derivative of covariance
        // matrix vector w.r.t. to vertical lengthscale and derivative of
        // covariance matrix vector w.r.t. to lengthscale
        K_tiles.resize(n_tiles * n_tiles);
        grad_v_tiles.resize(n_tiles * n_tiles);
        grad_l_tiles.resize(n_tiles * n_tiles);
        for (std::size_t i = 0; i < n_tiles; i++)
        {
            for (std::size_t j = 0; j <= i; j++)
            {
                hpx::shared_future<std::vector<double>> cov_dists =
                    hpx::async(
                        hpx::annotated_function(&compute_cov_dist_vec,
                                                "assemble_cov_dist"),
                        i,
                        j,
                        n_tile_size,
                        n_regressors,
                        sek_params,
                        training_input);

                K_tiles[i * n_tiles + j] = hpx::dataflow(
                    hpx::annotated_function(
                        hpx::unwrapping(&gen_tile_covariance_opt),
                        "assemble_K"),
                    i,
                    j,
                    n_tile_size,
                    n_regressors,
                    sek_params,
                    cov_dists);

                grad_v_tiles[i * n_tiles + j] =
                    hpx::dataflow(hpx::annotated_function(
                                      hpx::unwrapping(&gen_tile_grad_v),
                                      "assemble_gradv"),
                                  i,
                                  j,
                                  n_tile_size,
                                  n_regressors,
                                  sek_params,
                                  cov_dists);

                grad_l_tiles[i * n_tiles + j] =
                    hpx::dataflow(hpx::annotated_function(
                                      hpx::unwrapping(&gen_tile_grad_l),
                                      "assemble_gradl"),
                                  i,
                                  j,
                                  n_tile_size,
                                  n_regressors,
                                  sek_params,
                                  cov_dists);

                if (i != j)
                {
                    grad_v_tiles[j * n_tiles + i] = hpx::dataflow(
                        hpx::annotated_function(
                            hpx::unwrapping(&gen_tile_grad_v_trans),
                            "assemble_gradv_t"),
                        n_tile_size,
                        grad_v_tiles[i * n_tiles + j]);

                    grad_l_tiles[j * n_tiles + i] = hpx::dataflow(
                        hpx::annotated_function(
                            hpx::unwrapping(&gen_tile_grad_l_trans),
                            "assemble_gradl_t"),
                        n_tile_size,
                        grad_l_tiles[i * n_tiles + j]);
                }
            }
        }
        // Assemble placeholder matrix for K^-1 * (I - y*y^T*K^-1)
        grad_K_tiles.resize(n_tiles * n_tiles);
        for (std::size_t i = 0; i < n_tiles; i++)
        {
            for (std::size_t j = 0; j < n_tiles; j++)
            {
                grad_K_tiles[i * n_tiles + j] =
                    hpx::async(hpx::annotated_function(&gen_tile_identity,
                                                       "assemble_tiled"),
                               i,
                               j,
                               n_tile_size);
            }
        }
        // Assemble alpha
        alpha_tiles.resize(n_tiles);
        for (std::size_t i = 0; i < n_tiles; i++)
        {
            alpha_tiles[i] = hpx::async(
                hpx::annotated_function(&gen_tile_zeros, "assemble_tiled"),
                n_tile_size);
        }
        // Assemble placeholder matrix for K^-1
        grad_I_tiles.resize(n_tiles * n_tiles);
        for (std::size_t i = 0; i < n_tiles; i++)
        {
            for (std::size_t j = 0; j < n_tiles; j++)
            {
                grad_I_tiles[i * n_tiles + j] = hpx::async(
                    hpx::annotated_function(&gen_tile_identity,
                                            "assemble_identity_matrix"),
                    i,
                    j,
                    n_tile_size);
            }
        }

        //////////////////////////////////////////////////////////////////////////////
        // Cholesky decomposition
        right_looking_cholesky_tiled(target.cublas_executors, K_tiles, n_tile_size, n_tiles);
        // Compute K^-1 through L*L^T*X = I
        forward_solve_tiled_matrix(target.cublas_executors, K_tiles, grad_I_tiles, n_tile_size, n_tile_size, n_tiles, n_tiles);
        backward_solve_tiled_matrix(target.cublas_executors, K_tiles, grad_I_tiles, n_tile_size, n_tile_size, n_tiles, n_tiles);

        // Triangular solve K_NxN * alpha = y
        // forward_solve_tiled(grad_I_tiles, alpha_tiles, n_tile_size,
        // n_tiles); backward_solve_tiled(grad_I_tiles, alpha_tiles,
        // n_tile_size, n_tiles);

        // inv(K)*y
        compute_gemm_of_invK_y(target.cublas_executors, grad_I_tiles, y_tiles, alpha_tiles, n_tile_size, n_tiles);

        // Compute loss
        compute_loss_tiled(target.cublas_executors, K_tiles, alpha_tiles, y_tiles, loss_value, n_tile_size, n_tiles);
        losses[iter] = loss_value.get();

        // Compute I-y*y^T*inv(K) -> NxN matrix
        // update_grad_K_tiled(grad_K_tiles, y_tiles, alpha_tiles,
        // n_tile_size, n_tiles);

        // Compute K^-1 *(I - y*y^T*K^-1)
        // forward_solve_tiled_matrix(K_tiles, grad_K_tiles, n_tile_size,
        // n_tile_size, n_tiles, n_tiles);
        // backward_solve_tiled_matrix(K_tiles, grad_K_tiles, n_tile_size,
        // n_tile_size, n_tiles, n_tiles);

        // Update the hyperparameters
        if (trainable_params[0])
        {  // lengthscale
            sek_params.lengthscale = update_lengthscale(grad_I_tiles, grad_l_tiles, alpha_tiles, sek_params, adam_params, n_tile_size, n_tiles, m_T, v_T, beta1_T, beta2_T, 0);
        }
        if (trainable_params[1])
        {  // vertical_lengthscale
            sek_params.vertical_lengthscale = update_vertical_lengthscale(grad_I_tiles, grad_v_tiles, alpha_tiles, sek_params, adam_params, n_tile_size, n_tiles, m_T, v_T, beta1_T, beta2_T, 0);
        }
        if (trainable_params[2])
        {  // noise_variance
            sek_params.noise_variance = update_noise_variance(grad_I_tiles, alpha_tiles, sek_params, adam_params, n_tile_size, n_tiles, m_T, v_T, beta1_T, beta2_T, iter);
        }
    }
    // Update hyperparameter attributes in Gaussian process model
    // Return losses
    return hpx::async([losses]()
                      { return losses; }); */
    return hpx::shared_future<std::vector<double>>();
}

hpx::shared_future<double>
optimize_step(const std::vector<double> &training_input,
              const std::vector<double> &training_output,
              const std::size_t n_tiles,
              const std::size_t n_tile_size,
              const std::size_t n_regressors,
              const std::size_t iter,
              const gpxpy_hyper::SEKParams &sek_params,
              const std::vector<bool> trainable_params,
              const gpxpy_hyper::AdamParams &adam_params,
              gpxpy::CUDA_GPU &gpu)
{
    /* // declare tiled future data structures
    std::vector<hpx::shared_future<std::vector<double>>> K_tiles;
    std::vector<hpx::shared_future<std::vector<double>>> grad_v_tiles;
    std::vector<hpx::shared_future<std::vector<double>>> grad_l_tiles;
    std::vector<hpx::shared_future<std::vector<double>>> grad_K_tiles;
    std::vector<hpx::shared_future<std::vector<double>>> grad_I_tiles;
    std::vector<hpx::shared_future<std::vector<double>>> alpha_tiles;
    std::vector<hpx::shared_future<std::vector<double>>> y_tiles;
    // data holders for Adam
    std::vector<hpx::shared_future<double>> m_T;
    std::vector<hpx::shared_future<double>> v_T;
    std::vector<hpx::shared_future<double>> beta1_T;
    std::vector<hpx::shared_future<double>> beta2_T;
    // data holder for loss
    hpx::shared_future<double> loss_value;
    // make shared future
    for (std::size_t i = 0; i < 3; i++)
    {
        hpx::shared_future<double> m =
            hpx::make_ready_future(adam_params.M_T[i]);  //.share();
        m_T.push_back(m);
        hpx::shared_future<double> v =
            hpx::make_ready_future(adam_params.V_T[i]);  //.share();
        v_T.push_back(v);
    }

    // Assemble beta1_t and beta2_t
    beta1_T.resize(1);
    beta1_T[0] =
        hpx::async(hpx::annotated_function(&gen_beta_T, "assemble_tiled"),
                   iter + 1,
                   adam_params.beta1);

    beta2_T.resize(1);
    beta2_T[0] =
        hpx::async(hpx::annotated_function(&gen_beta_T, "assemble_tiled"),
                   iter + 1,
                   adam_params.beta1);

    // Assemble covariance matrix vector
    K_tiles.resize(n_tiles * n_tiles);
    for (std::size_t i = 0; i < n_tiles; i++)
    {
        for (std::size_t j = 0; j <= i; j++)
        {
            K_tiles[i * n_tiles + j] =
                hpx::async(hpx::annotated_function(&gen_tile_covariance,
                                                   "assemble_tiled"),
                           i,
                           j,
                           n_tile_size,
                           n_regressors,
                           sek_params,
                           training_input);
        }
    }
    // Assemble derivative of covariance matrix vector w.r.t. to vertical
    // lengthscale
    grad_v_tiles.resize(n_tiles * n_tiles);
    for (std::size_t i = 0; i < n_tiles; i++)
    {
        for (std::size_t j = 0; j < n_tiles; j++)
        {
            grad_v_tiles[i * n_tiles + j] = hpx::async(
                hpx::annotated_function(&gen_tile_grad_v, "assemble_tiled"),
                i,
                j,
                n_tile_size,
                n_regressors,
                sek_params,
                training_input);
        }
    }
    // Assemble derivative of covariance matrix vector w.r.t. to lengthscale
    grad_l_tiles.resize(n_tiles * n_tiles);
    for (std::size_t i = 0; i < n_tiles; i++)
    {
        for (std::size_t j = 0; j < n_tiles; j++)
        {
            grad_l_tiles[i * n_tiles + j] = hpx::async(
                hpx::annotated_function(&gen_tile_grad_l, "assemble_tiled"),
                i,
                j,
                n_tile_size,
                n_regressors,
                sek_params,
                training_input);
        }
    }
    // Assemble matrix that will be multiplied with derivates
    grad_K_tiles.resize(n_tiles * n_tiles);
    for (std::size_t i = 0; i < n_tiles; i++)
    {
        for (std::size_t j = 0; j < n_tiles; j++)
        {
            grad_K_tiles[i * n_tiles + j] =
                hpx::async(hpx::annotated_function(&gen_tile_identity,
                                                   "assemble_tiled"),
                           i,
                           j,
                           n_tile_size);
        }
    }
    // Assemble alpha
    alpha_tiles.resize(n_tiles);
    for (std::size_t i = 0; i < n_tiles; i++)
    {
        alpha_tiles[i] = hpx::async(
            hpx::annotated_function(&gen_tile_output, "assemble_tiled"), i, n_tile_size, training_output);
    }
    // Assemble y
    y_tiles.resize(n_tiles);
    for (std::size_t i = 0; i < n_tiles; i++)
    {
        y_tiles[i] = hpx::async(
            hpx::annotated_function(&gen_tile_output, "assemble_tiled"), i, n_tile_size, training_output);
    }
    // Assemble placeholder matrix for K^-1
    grad_I_tiles.resize(n_tiles * n_tiles);
    for (std::size_t i = 0; i < n_tiles; i++)
    {
        for (std::size_t j = 0; j < n_tiles; j++)
        {
            grad_I_tiles[i * n_tiles + j] = hpx::async(
                hpx::annotated_function(&gen_tile_identity,
                                        "assemble_identity_matrix"),
                i,
                j,
                n_tile_size);
        }
    }

    // Cholesky decomposition
    right_looking_cholesky_tiled(target.cublas_executors, K_tiles, n_tile_size, n_tiles);

    // Triangular solve K_NxN * alpha = y
    forward_solve_tiled(target.cublas_executors, K_tiles, alpha_tiles, n_tile_size, n_tiles);
    backward_solve_tiled(target.cublas_executors, K_tiles, alpha_tiles, n_tile_size, n_tiles);

    // Compute K^-1 through L*L^T*X = I
    forward_solve_tiled_matrix(target.cublas_executors, K_tiles, grad_I_tiles, n_tile_size, n_tile_size, n_tiles, n_tiles);
    backward_solve_tiled_matrix(target.cublas_executors, K_tiles, grad_I_tiles, n_tile_size, n_tile_size, n_tiles, n_tiles);

    // Compute loss
    compute_loss_tiled(target.cublas_executors, K_tiles, alpha_tiles, y_tiles, loss_value, n_tile_size, n_tiles);

    // // Fill I-y*y^T*inv(K)
    // update_grad_K_tiled(grad_K_tiles, y_tiles, alpha_tiles, n_tile_size,
    // n_tiles);

    // // Compute K^-1 * (I-y*y^T*K^-1)
    // forward_solve_tiled_matrix(K_tiles, grad_K_tiles, n_tile_size,
    // n_tile_size, n_tiles, n_tiles); backward_solve_tiled_matrix(K_tiles,
    // grad_K_tiles, n_tile_size, n_tile_size, n_tiles, n_tiles);

    // Update the hyperparameters
    if (trainable_params[0])
    {  // lengthscale
        sek_params.lengthscale = update_lengthscale(grad_I_tiles, grad_l_tiles, alpha_tiles, sek_params, adam_params, n_tile_size, n_tiles, m_T, v_T, beta1_T, beta2_T, 0);
    }
    if (trainable_params[1])
    {  // vertical_lengthscale
        sek_params.vertical_lengthscale = update_vertical_lengthscale(grad_I_tiles, grad_v_tiles, alpha_tiles, sek_params, adam_params, n_tile_size, n_tiles, m_T, v_T, beta1_T, beta2_T, 0);
    }
    if (trainable_params[2])
    {  // noise_variance
        sek_params.noise_variance = update_noise_variance(grad_I_tiles, alpha_tiles, sek_params, adam_params, n_tile_size, n_tiles, m_T, v_T, beta1_T, beta2_T, iter);
    }

    // Update hyperparameter attributes (first and second moment) for Adam
    for (std::size_t i = 0; i < 3; i++)
    {
        adam_params.M_T[i] = m_T[i].get();
        adam_params.V_T[i] = v_T[i].get();
    }

    // Return loss value
    double loss = loss_value.get();
    return hpx::async([loss]()
                      { return loss; }); */
    return hpx::shared_future<double>();
}

hpx::shared_future<std::vector<std::vector<double>>>
cholesky(const std::vector<double> &h_training_input,
         const std::size_t n_tiles,
         const std::size_t n_tile_size,
         const std::size_t n_regressors,
         const gpxpy_hyper::SEKParams sek_params,
         gpxpy::CUDA_GPU &gpu)
{
#if GPRAT_CHOLESKY_STEPS
    auto cholesky_step_ra_timer = apex::start("cholesky_step ressource allocation");
#endif
    gpu.create();
    hipsolverHandle_t cusolver = create_cusolver_handle();

#if GPRAT_CHOLESKY_STEPS
    apex::stop(cholesky_step_ra_timer);
#endif

#if GPRAT_CHOLESKY_STEPS || GPRAT_ASSEMBLY_ONLY
    auto cholesky_step_assembly_timer = apex::start("cholesky_step assembly");
#endif

    double *d_training_input = copy_to_device(h_training_input, gpu);
    // Assemble tiled covariance matrix on GPU.
    std::vector<hpx::shared_future<double *>> d_tiles = assemble_tiled_covariance_matrix(d_training_input, n_tiles, n_tile_size, n_regressors, sek_params, gpu);

#if GPRAT_ASSEMBLY_ONLY
    hpx::wait_all(d_tiles);
    apex::stop(cholesky_step_assembly_timer);
    return hpx::make_ready_future(std::vector<std::vector<double>>());
#endif
#if GPRAT_CHOLESKY_STEPS
    hpx::wait_all(d_tiles);
    apex::stop(cholesky_step_assembly_timer);
    auto cholesky_step_cholesky_timer = apex::start("cholesky_step cholesky");
#endif

    // Compute Tiled Cholesky decomposition on device
    right_looking_cholesky_tiled(d_tiles, n_tile_size, n_tiles, gpu, cusolver);

#if GPRAT_CHOLESKY_STEPS
    hpx::wait_all(d_tiles);
    apex::stop(cholesky_step_cholesky_timer);
    auto cholesky_step_copyback_timer = apex::start("cholesky_step copyback");
#endif

    // Copy tiled matrix to host
    std::vector<std::vector<double>> h_tiles = move_lower_tiled_matrix_to_host(d_tiles, n_tile_size, n_tiles, gpu);

#if GPRAT_CHOLESKY_STEPS
    apex::stop(cholesky_step_copyback_timer);
    auto cholesky_step_rd_timer = apex::start("cholesky_step ressource destroy");
#endif

    hipFree(d_training_input);
    destroy(cusolver);
    gpu.destroy();

#if GPRAT_CHOLESKY_STEPS
    apex::stop(cholesky_step_rd_timer);
#endif

    return hpx::make_ready_future(h_tiles);
}

}  // end of namespace gpu
